
#include <hip/hip_runtime.h>
#include <cstdio>

__global__  // __global__ functions, or "kernels", execute on the device
void hello_kernel() {
    printf("Hello world from the device!\n");
}

int main() {

    // greet from the host
    printf("Hello world from the host!\n");

    // launch a kernel with a single thread to greet from the device
    hello_kernel<<<1,1>>>();
    
    // wait for the device to finish so that we see the message
    hipDeviceSynchronize();

    return 0;
}
