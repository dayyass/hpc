
#include <hip/hip_runtime.h>
#include <cstdio>

__global__
void kernel(int rank, int size) {
    printf("Hello world from the device, rank %d out of %d processors\n", rank, size);
}

extern void launch_kernel(int rank, int size) {
    kernel<<<1,1>>>(rank, size);
    hipDeviceSynchronize();
}
